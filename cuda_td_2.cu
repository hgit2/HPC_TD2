#include "hip/hip_runtime.h"
#include "wb.h"

#define wbCheck(stmt)\
    do {\
        hipError_t err = stmt; \
        if (err != hipSuccess) { \
            wbLog(ERROR, "Failed to run stmt ", #stmt);\
            wbLog(ERROR, "Got CUDA error ... ", hipGetErrorString(err));\
            return -1;\
        }\
    } while (0)

#define CHANNELS 3
__global__ void colorConvert(float * grayImg, float * rgbImg, int width, int height);
    
//@@ INSERT CODE HERE
int main(int argc, char *argv[]) {
    wbArg_t args;
    int imageChannels;
    int imageWidth;
    int imageHeight;
    char *inputImageFile;
    wbImage_t inputImage;
    wbImage_t outputImage;
    float *hostInputImageData;
    float *hostOutputImageData;
    float *deviceInputImageData;
    float *deviceOutputImageData;
    args = wbArg_read(argc, argv); /* parse the input arguments */
    inputImageFile = wbArg_getInputFile(args, 0);
    inputImage = wbImport(inputImageFile);
    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    // For this lab the value is always 3
    imageChannels = wbImage_getChannels(inputImage);
    // Since the image is monochromatic, it only contains one channel
    outputImage = wbImage_new(imageWidth, imageHeight, 1);
    hostInputImageData = wbImage_getData(inputImage);
    hostOutputImageData = wbImage_getData(outputImage);
    wbTime_start(GPU, "Doing GPU Computation (memory + compute)");
    wbTime_start(GPU, "Doing GPU memory allocation");
    hipMalloc((void **)&deviceInputImageData,imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **)&deviceOutputImageData,imageWidth * imageHeight * sizeof(float));
    wbTime_stop(GPU, "Doing GPU memory allocation");
    wbTime_start(Copy, "Copying data to the GPU");
    hipMemcpy(deviceInputImageData, hostInputImageData,imageWidth * imageHeight * imageChannels * sizeof(float),hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying data to the GPU");
    ///////////////////////////////////////////////////////
    wbTime_start(Compute, "Doing the computation on the GPU");

    //@@ INSERT CODE HERE to convert colored image to gray
		dim3 DimGrid( (imageHeight - 1)/16, (imageWidth - 1)/16+1, 1 );
		dim3 DimBlock(16,16,1);
		colorConvert <<< DimGrid, DimBlock >>> (deviceOutputImageData, deviceInputImageData, imageWidth, imageHeight);


    wbTime_stop(Compute, "Doing the computation on the GPU");
    ///////////////////////////////////////////////////////
    wbTime_start(Copy, "Copying data from the GPU");
    hipMemcpy(hostOutputImageData, deviceOutputImageData,
    imageWidth * imageHeight * sizeof(float),
    hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying data from the GPU");
    wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");
    
    
    unsigned char grayScale[imageHeight][imageWidth];
 
    int i, j;
    for (j = 0; j < imageHeight; ++j) {
        for (i = 0; i < imageWidth; ++i) {
            grayScale[j][i] = ceil(hostOutputImageData[i + j * imageWidth] * 255.0);
        }
    }
 
    FILE *fp = fopen("grayImageFile.ppm", "wb"); /* b - binary mode */
    fprintf(fp, "P5\n%d %d\n255\n", imageWidth, imageHeight);
    fwrite(grayScale, sizeof(grayScale), 1, fp);
    fclose(fp);
    
    
    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    wbImage_delete(outputImage);
    wbImage_delete(inputImage);
    return 0;
}

// define colorConvert
		__global__ void colorConvert(float * grayImg, float * rgbImg, int width, int height){
			int x = threadIdx.x + blockDim.x * blockIdx.x ;
			int y = threadIdx.y + blockDim.y * blockIdx.y ; 
			if(y<width && x< height){
				int grayOffset = y*width + x;
				int rgbOffset = grayOffset*CHANNELS;
				float r = rgbImg[rgbOffset];
				float g = rgbImg[rgbOffset+2];
				float b = rgbImg[rgbOffset+3];
				grayImg[grayOffset]= 0.21f*r + 0.71f*g + 0.07f*b;
			}
		}
