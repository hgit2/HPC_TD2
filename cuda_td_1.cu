#include "hip/hip_runtime.h"
#include "wb.h"

void vecAdd(float *in1, float *in2, float *out, int len) {
	vecAddKernel <<< ceil(len/256.0),256 >>> (in1 , in2 , out , len );
}

__global__ void vecAddKernel( float *in1, float *in2, float *out, int len){
	int i = threadIdx.x+blockDim.x∗blockIdx.x;
	if(i< len){
		out[i]=in1[i]+in2[i];
	}
}

int main(int argc, char **argv) {
    wbArg_t args;
    int inputLength;
    float *hostInput1;
    float *hostInput2;
    float *hostOutput;
    args = wbArg_read(argc, argv);
    wbTime_start(Generic, "Importing data and creating memory on host");

		// Rappel : CPU=host
    hostInput1 =
    (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 =
    (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *)malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");
    wbLog(TRACE, "The input length is ", inputLength);
    wbTime_start(GPU, "Allocating GPU memory.");

    //@@ Allocate GPU memory here
		hipMalloc(( void ∗∗ ) &deviceInput1 , inputlength * sizeof(float));		
		hipMalloc(( void ∗∗ ) &deviceInput2 , inputlength * sizeof(float));
		hipMalloc(( void ∗∗ ) &deviceOutput , inputlength * sizeof(float));

    wbTime_stop(GPU, "Allocating GPU memory.");
    wbTime_start(GPU, "Copying input memory to the GPU.");

    //@@ Copy memory to the GPU here
		hipMemcpy( deviceInput1 , hostInput1 , inputlength * sizeof(float) , hipMemcpyHostToDevice );
		hipMemcpy( deviceInput2 , hostInput2 , inputlength  * sizeof(float), hipMemcpyHostToDevice );

    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
		// cf fonction vecadd

    wbTime_start(Compute, "Performing CUDA computation");

    //@@ Launch the GPU Kernel here
		vecAdd(deviceInput1, deviceInput2, deviceOutput, inputlength)

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    wbTime_start(Copy, "Copying output memory to the CPU");

    //@@ Copy the GPU memory back to the CPU here
		hipMemcpy( hostOutput , deviceOutput , inputlength * sizeof(float) , hipMemcpyDeviceToHost);

    wbTime_stop(Copy, "Copying output memory to the CPU");
    wbTime_start(GPU, "Freeing GPU Memory");

    //@@ Free the GPU memory here
		hipFree( deviceInput1 ); 
		hipFree( deviceInput2 );
		hipFree( deviceOutput );

    wbTime_stop(GPU, "Freeing GPU Memory");
    wbSolution(args, hostOutput, inputLength);
    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}
